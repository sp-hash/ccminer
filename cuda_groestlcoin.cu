#include "hip/hip_runtime.h"
// Auf Groestlcoin spezialisierte Version von Groestl inkl. Bitslice

#include <stdio.h>
#include <memory.h>

#include "cuda_helper.h"
#include <host_defines.h>

// globaler Speicher für alle HeftyHashes aller Threads
__constant__ uint32_t pTarget[8]; // Single GPU
extern uint32_t *d_resultNonce[8];

__constant__ uint32_t groestlcoin_gpu_msg[32];

// 64 Register Variante für Compute 3.0
#include "groestl_functions_quad.cu"
#include "bitslice_transformations_quad.cu"

#define SWAB32(x) cuda_swab32(x)

__global__ __launch_bounds__(256, 4)
void groestlcoin_gpu_hash_quad(int threads, uint32_t startNounce, uint32_t *resNounce)
{
    // durch 4 dividieren, weil jeweils 4 Threads zusammen ein Hash berechnen
    int thread = (blockDim.x * blockIdx.x + threadIdx.x) / 4;
    if (thread < threads)
    {
        // GROESTL
        uint32_t paddedInput[8];
#pragma unroll 8
        for(int k=0;k<8;k++) paddedInput[k] = groestlcoin_gpu_msg[4*k+threadIdx.x%4];

        uint32_t nounce = startNounce + thread;
        if ((threadIdx.x % 4) == 3)
            paddedInput[4] = SWAB32(nounce);  // 4*4+3 = 19

        uint32_t msgBitsliced[8];
        to_bitslice_quad(paddedInput, msgBitsliced);

        uint32_t state[8];
        for (int round=0; round<2; round++)
        {
            groestl512_progressMessage_quad(state, msgBitsliced);

            if (round < 1)
            {
                // Verkettung zweier Runden inclusive Padding.
                msgBitsliced[ 0] = __byte_perm(state[ 0], 0x00800100, 0x4341 + ((threadIdx.x%4)==3)*0x2000);
                msgBitsliced[ 1] = __byte_perm(state[ 1], 0x00800100, 0x4341);
                msgBitsliced[ 2] = __byte_perm(state[ 2], 0x00800100, 0x4341);
                msgBitsliced[ 3] = __byte_perm(state[ 3], 0x00800100, 0x4341);
                msgBitsliced[ 4] = __byte_perm(state[ 4], 0x00800100, 0x4341);
                msgBitsliced[ 5] = __byte_perm(state[ 5], 0x00800100, 0x4341);
                msgBitsliced[ 6] = __byte_perm(state[ 6], 0x00800100, 0x4341);
                msgBitsliced[ 7] = __byte_perm(state[ 7], 0x00800100, 0x4341 + ((threadIdx.x%4)==0)*0x0010);
            }
        }

        // Nur der erste von jeweils 4 Threads bekommt das Ergebns-Hash
        uint32_t out_state[16];
        from_bitslice_quad(state, out_state);
        
        if (threadIdx.x % 4 == 0)
        {
			if (cuda_hashisbelowtarget(out_state, pTarget))
				resNounce[0] = nounce;
		}
    }
}

// Setup-Funktionen
__host__ void groestlcoin_cpu_init(int thr_id, int threads)
{
    hipSetDevice(device_map[thr_id]);

    // Speicher für Gewinner-Nonce belegen
    hipMalloc(&d_resultNonce[thr_id], sizeof(uint32_t)); 
}

__host__ void groestlcoin_cpu_setBlock(int thr_id, void *data, void *pTargetIn)
{
    // Nachricht expandieren und setzen
    uint32_t msgBlock[32];

    memset(msgBlock, 0, sizeof(uint32_t) * 32);
    memcpy(&msgBlock[0], data, 80);

    // Erweitere die Nachricht auf den Nachrichtenblock (padding)
    // Unsere Nachricht hat 80 Byte
    msgBlock[20] = 0x80;
    msgBlock[31] = 0x01000000;

    // groestl512 braucht hierfür keinen CPU-Code (die einzige Runde wird
    // auf der GPU ausgeführt)

    // Blockheader setzen (korrekte Nonce und Hefty Hash fehlen da drin noch)
    hipMemcpyToSymbol(HIP_SYMBOL( groestlcoin_gpu_msg),
                        msgBlock,
                        128);

    hipMemset(d_resultNonce[thr_id], 0xFF, sizeof(uint32_t));
    hipMemcpyToSymbol(HIP_SYMBOL( pTarget),
                        pTargetIn,
                        sizeof(uint32_t) * 8 );
}

__host__ void groestlcoin_cpu_hash(int thr_id, int threads, uint32_t startNounce, void *outputHashes, uint32_t *nounce)
{
    int threadsperblock = 256;

    // Compute 3.0 benutzt die registeroptimierte Quad Variante mit Warp Shuffle
    // mit den Quad Funktionen brauchen wir jetzt 4 threads pro Hash, daher Faktor 4 bei der Blockzahl
    int factor = 4;

        // berechne wie viele Thread Blocks wir brauchen
    dim3 grid(factor*((threads + threadsperblock-1)/threadsperblock));
    dim3 block(threadsperblock);

    // Größe des dynamischen Shared Memory Bereichs
    size_t shared_size = 0;

    hipMemset(d_resultNonce[thr_id], 0xFF, sizeof(uint32_t));
    groestlcoin_gpu_hash_quad<<<grid, block, shared_size>>>(threads, startNounce, d_resultNonce[thr_id]);

    // Strategisches Sleep Kommando zur Senkung der CPU Last
    MyStreamSynchronize(NULL, 0, thr_id);

    hipMemcpy(nounce, d_resultNonce[thr_id], sizeof(uint32_t), hipMemcpyDeviceToHost);
}
