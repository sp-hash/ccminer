#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <memory.h>

#include "cuda_helper.h"

// aus cpu-miner.c
extern int device_map[8];

// Take a look at: https://www.schneier.com/skein1.3.pdf

#define SHL(x, n)			((x) << (n))
#define SHR(x, n)			((x) >> (n))

__constant__ uint32_t pTarget[8];
static uint32_t *d_nonce[8];

#if __CUDA_ARCH__ >= 320
__device__
uint64_t skein_rotl64(const uint64_t x, const int offset)
{
	uint64_t res;
	if(offset<32)
	{
		asm("{\n\t"
			".reg .u32 tl,th,vl,vh;\n\t"
			"mov.b64 {tl,th}, %1;\n\t"
			"shf.l.wrap.b32 vl, tl, th, %2;\n\t"
			"shf.l.wrap.b32 vh, th, tl, %2;\n\t"
			"mov.b64 %0, {vh,vl};\n\t"
			"}"
			: "=l"(res) : "l"(x) , "r"(offset)
			);
	} else
	{
		asm("{\n\t"
			".reg .u32 tl,th,vl,vh;\n\t"
			"mov.b64 {tl,th}, %1;\n\t"
			"shf.l.wrap.b32 vl, tl, th, %2;\n\t"
			"shf.l.wrap.b32 vh, th, tl, %2;\n\t"
			"mov.b64 %0, {vl,vh};\n\t"
			"}"
			: "=l"(res) : "l"(x) , "r"(offset)
			);
	}
	return res;
}
#undef ROTL64
#define ROTL64 skein_rotl64
#endif

/*
 * M9_ ## s ## _ ## i  evaluates to s+i mod 9 (0 <= s <= 18, 0 <= i <= 7).
 */

#define M9_0_0    0
#define M9_0_1    1
#define M9_0_2    2
#define M9_0_3    3
#define M9_0_4    4
#define M9_0_5    5
#define M9_0_6    6
#define M9_0_7    7

#define M9_1_0    1
#define M9_1_1    2
#define M9_1_2    3
#define M9_1_3    4
#define M9_1_4    5
#define M9_1_5    6
#define M9_1_6    7
#define M9_1_7    8

#define M9_2_0    2
#define M9_2_1    3
#define M9_2_2    4
#define M9_2_3    5
#define M9_2_4    6
#define M9_2_5    7
#define M9_2_6    8
#define M9_2_7    0

#define M9_3_0    3
#define M9_3_1    4
#define M9_3_2    5
#define M9_3_3    6
#define M9_3_4    7
#define M9_3_5    8
#define M9_3_6    0
#define M9_3_7    1

#define M9_4_0    4
#define M9_4_1    5
#define M9_4_2    6
#define M9_4_3    7
#define M9_4_4    8
#define M9_4_5    0
#define M9_4_6    1
#define M9_4_7    2

#define M9_5_0    5
#define M9_5_1    6
#define M9_5_2    7
#define M9_5_3    8
#define M9_5_4    0
#define M9_5_5    1
#define M9_5_6    2
#define M9_5_7    3

#define M9_6_0    6
#define M9_6_1    7
#define M9_6_2    8
#define M9_6_3    0
#define M9_6_4    1
#define M9_6_5    2
#define M9_6_6    3
#define M9_6_7    4

#define M9_7_0    7
#define M9_7_1    8
#define M9_7_2    0
#define M9_7_3    1
#define M9_7_4    2
#define M9_7_5    3
#define M9_7_6    4
#define M9_7_7    5

#define M9_8_0    8
#define M9_8_1    0
#define M9_8_2    1
#define M9_8_3    2
#define M9_8_4    3
#define M9_8_5    4
#define M9_8_6    5
#define M9_8_7    6

#define M9_9_0    0
#define M9_9_1    1
#define M9_9_2    2
#define M9_9_3    3
#define M9_9_4    4
#define M9_9_5    5
#define M9_9_6    6
#define M9_9_7    7

#define M9_10_0   1
#define M9_10_1   2
#define M9_10_2   3
#define M9_10_3   4
#define M9_10_4   5
#define M9_10_5   6
#define M9_10_6   7
#define M9_10_7   8

#define M9_11_0   2
#define M9_11_1   3
#define M9_11_2   4
#define M9_11_3   5
#define M9_11_4   6
#define M9_11_5   7
#define M9_11_6   8
#define M9_11_7   0

#define M9_12_0   3
#define M9_12_1   4
#define M9_12_2   5
#define M9_12_3   6
#define M9_12_4   7
#define M9_12_5   8
#define M9_12_6   0
#define M9_12_7   1

#define M9_13_0   4
#define M9_13_1   5
#define M9_13_2   6
#define M9_13_3   7
#define M9_13_4   8
#define M9_13_5   0
#define M9_13_6   1
#define M9_13_7   2

#define M9_14_0   5
#define M9_14_1   6
#define M9_14_2   7
#define M9_14_3   8
#define M9_14_4   0
#define M9_14_5   1
#define M9_14_6   2
#define M9_14_7   3

#define M9_15_0   6
#define M9_15_1   7
#define M9_15_2   8
#define M9_15_3   0
#define M9_15_4   1
#define M9_15_5   2
#define M9_15_6   3
#define M9_15_7   4

#define M9_16_0   7
#define M9_16_1   8
#define M9_16_2   0
#define M9_16_3   1
#define M9_16_4   2
#define M9_16_5   3
#define M9_16_6   4
#define M9_16_7   5

#define M9_17_0   8
#define M9_17_1   0
#define M9_17_2   1
#define M9_17_3   2
#define M9_17_4   3
#define M9_17_5   4
#define M9_17_6   5
#define M9_17_7   6

#define M9_18_0   0
#define M9_18_1   1
#define M9_18_2   2
#define M9_18_3   3
#define M9_18_4   4
#define M9_18_5   5
#define M9_18_6   6
#define M9_18_7   7

/*
 * M3_ ## s ## _ ## i  evaluates to s+i mod 3 (0 <= s <= 18, 0 <= i <= 1).
 */

#define M3_0_0    0
#define M3_0_1    1
#define M3_1_0    1
#define M3_1_1    2
#define M3_2_0    2
#define M3_2_1    0
#define M3_3_0    0
#define M3_3_1    1
#define M3_4_0    1
#define M3_4_1    2
#define M3_5_0    2
#define M3_5_1    0
#define M3_6_0    0
#define M3_6_1    1
#define M3_7_0    1
#define M3_7_1    2
#define M3_8_0    2
#define M3_8_1    0
#define M3_9_0    0
#define M3_9_1    1
#define M3_10_0   1
#define M3_10_1   2
#define M3_11_0   2
#define M3_11_1   0
#define M3_12_0   0
#define M3_12_1   1
#define M3_13_0   1
#define M3_13_1   2
#define M3_14_0   2
#define M3_14_1   0
#define M3_15_0   0
#define M3_15_1   1
#define M3_16_0   1
#define M3_16_1   2
#define M3_17_0   2
#define M3_17_1   0
#define M3_18_0   0
#define M3_18_1   1

#define XCAT(x, y)     XCAT_(x, y)
#define XCAT_(x, y)    x ## y

#define SKBI(k, s, i)   XCAT(k, XCAT(XCAT(XCAT(M9_, s), _), i))
#define SKBT(t, s, v)   XCAT(t, XCAT(XCAT(XCAT(M3_, s), _), v))

#define TFBIG_KINIT(k0, k1, k2, k3, k4, k5, k6, k7, k8, t0, t1, t2) { \
		k8 = ((k0 ^ k1) ^ (k2 ^ k3)) ^ ((k4 ^ k5) ^ (k6 ^ k7)) \
			^ SPH_C64(0x1BD11BDAA9FC1A22); \
		t2 = t0 ^ t1; \
	}

#define TFBIG_ADDKEY(w0, w1, w2, w3, w4, w5, w6, w7, k, t, s) { \
		w0 = (w0 + SKBI(k, s, 0)); \
		w1 = (w1 + SKBI(k, s, 1)); \
		w2 = (w2 + SKBI(k, s, 2)); \
		w3 = (w3 + SKBI(k, s, 3)); \
		w4 = (w4 + SKBI(k, s, 4)); \
		w5 = (w5 + SKBI(k, s, 5) + SKBT(t, s, 0)); \
		w6 = (w6 + SKBI(k, s, 6) + SKBT(t, s, 1)); \
		w7 = (w7 + SKBI(k, s, 7) + (uint64_t)s); \
	}

#define TFBIG_MIX(x0, x1, rc) { \
		x0 = x0 + x1; \
		x1 = ROTL64(x1, rc) ^ x0; \
	}

#define TFBIG_MIX8(w0, w1, w2, w3, w4, w5, w6, w7, rc0, rc1, rc2, rc3) { \
		TFBIG_MIX(w0, w1, rc0); \
		TFBIG_MIX(w2, w3, rc1); \
		TFBIG_MIX(w4, w5, rc2); \
		TFBIG_MIX(w6, w7, rc3); \
	}

#define TFBIG_4e(s)  { \
		TFBIG_ADDKEY(p[0], p[1], p[2], p[3], p[4], p[5], p[6], p[7], h, t, s); \
		TFBIG_MIX8(p[0], p[1], p[2], p[3], p[4], p[5], p[6], p[7], 46, 36, 19, 37); \
		TFBIG_MIX8(p[2], p[1], p[4], p[7], p[6], p[5], p[0], p[3], 33, 27, 14, 42); \
		TFBIG_MIX8(p[4], p[1], p[6], p[3], p[0], p[5], p[2], p[7], 17, 49, 36, 39); \
		TFBIG_MIX8(p[6], p[1], p[0], p[7], p[2], p[5], p[4], p[3], 44,  9, 54, 56); \
	}

#define TFBIG_4o(s)  { \
		TFBIG_ADDKEY(p[0], p[1], p[2], p[3], p[4], p[5], p[6], p[7], h, t, s); \
		TFBIG_MIX8(p[0], p[1], p[2], p[3], p[4], p[5], p[6], p[7], 39, 30, 34, 24); \
		TFBIG_MIX8(p[2], p[1], p[4], p[7], p[6], p[5], p[0], p[3], 13, 50, 10, 17); \
		TFBIG_MIX8(p[4], p[1], p[6], p[3], p[0], p[5], p[2], p[7], 25, 29, 39, 43); \
		TFBIG_MIX8(p[6], p[1], p[0], p[7], p[2], p[5], p[4], p[3],  8, 35, 56, 22); \
	}
__global__ 
void quark_skein512_gpu_hash_64(int threads, uint32_t startNounce, uint64_t * const __restrict__ g_hash, uint32_t *g_nonceVector)
{
	int thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		// Skein
		uint64_t p[8];
		uint64_t h0, h1, h2, h3, h4, h5, h6, h7, h8;
		uint64_t t0, t1, t2;

		uint32_t nounce = (g_nonceVector != NULL) ? g_nonceVector[thread] : (startNounce + thread);

		int hashPosition = nounce - startNounce;
		uint64_t *inpHash = &g_hash[8 * hashPosition];

		// Initialisierung
		h0 = 0x4903ADFF749C51CEull;
		h1 = 0x0D95DE399746DF03ull;
		h2 = 0x8FD1934127C79BCEull;
		h3 = 0x9A255629FF352CB1ull;
		h4 = 0x5DB62599DF6CA7B0ull;
		h5 = 0xEABE394CA9D5C3F4ull;
		h6 = 0x991112C71A75B523ull;
		h7 = 0xAE18A40B660FCC33ull;

		// 1. Runde -> etype = 480, ptr = 64, bcount = 0, data = msg		
#pragma unroll 8
		for(int i=0;i<8;i++)
			p[i] = inpHash[i];

		t0 = 64; // ptr
		t1 = 480ull << 55; // etype
		TFBIG_KINIT(h0, h1, h2, h3, h4, h5, h6, h7, h8, t0, t1, t2);
		TFBIG_4e(0);
		TFBIG_4o(1);
		TFBIG_4e(2);
		TFBIG_4o(3);
		TFBIG_4e(4);
		TFBIG_4o(5);
		TFBIG_4e(6);
		TFBIG_4o(7);
		TFBIG_4e(8);
		TFBIG_4o(9);
		TFBIG_4e(10);
		TFBIG_4o(11);
		TFBIG_4e(12);
		TFBIG_4o(13);
		TFBIG_4e(14);
		TFBIG_4o(15);
		TFBIG_4e(16);
		TFBIG_4o(17);
		TFBIG_ADDKEY(p[0], p[1], p[2], p[3], p[4], p[5], p[6], p[7], h, t, 18);

		h0 = inpHash[0] ^ p[0];
		h1 = inpHash[1] ^ p[1];
		h2 = inpHash[2] ^ p[2];
		h3 = inpHash[3] ^ p[3];
		h4 = inpHash[4] ^ p[4];
		h5 = inpHash[5] ^ p[5];
		h6 = inpHash[6] ^ p[6];
		h7 = inpHash[7] ^ p[7];

		// 2. Runde -> etype = 510, ptr = 8, bcount = 0, data = 0
#pragma unroll 8
		for(int i=0;i<8;i++)
			p[i] = 0;

		t0 = 8; // ptr
		t1 = 510ull << 55; // etype
		TFBIG_KINIT(h0, h1, h2, h3, h4, h5, h6, h7, h8, t0, t1, t2);
		TFBIG_4e(0);
		TFBIG_4o(1);
		TFBIG_4e(2);
		TFBIG_4o(3);
		TFBIG_4e(4);
		TFBIG_4o(5);
		TFBIG_4e(6);
		TFBIG_4o(7);
		TFBIG_4e(8);
		TFBIG_4o(9);
		TFBIG_4e(10);
		TFBIG_4o(11);
		TFBIG_4e(12);
		TFBIG_4o(13);
		TFBIG_4e(14);
		TFBIG_4o(15);
		TFBIG_4e(16);
		TFBIG_4o(17);
		TFBIG_ADDKEY(p[0], p[1], p[2], p[3], p[4], p[5], p[6], p[7], h, t, 18);

		// fertig
		uint64_t *outpHash = &g_hash[8 * hashPosition];

#pragma unroll 8
		for(int i=0;i<8;i++)
			outpHash[i] = p[i];
	}
}

__global__ 
void quark_skein512_gpu_hash_64_final(const int threads,const uint32_t startNounce, uint64_t * const __restrict__ g_hash, const uint32_t *g_nonceVector, uint32_t *d_nonce)
{
	int thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		// Skein
		uint64_t p[8];
		uint64_t h0, h1, h2, h3, h4, h5, h6, h7, h8;
		uint64_t t0, t1, t2;

		uint32_t nounce = (g_nonceVector != NULL) ? g_nonceVector[thread] : (startNounce + thread);

		int hashPosition = nounce - startNounce;
		uint64_t *inpHash = &g_hash[8 * hashPosition];

		// Initialisierung
		h0 = 0x4903ADFF749C51CEull;
		h1 = 0x0D95DE399746DF03ull;
		h2 = 0x8FD1934127C79BCEull;
		h3 = 0x9A255629FF352CB1ull;
		h4 = 0x5DB62599DF6CA7B0ull;
		h5 = 0xEABE394CA9D5C3F4ull;
		h6 = 0x991112C71A75B523ull;
		h7 = 0xAE18A40B660FCC33ull;

		// 1. Runde -> etype = 480, ptr = 64, bcount = 0, data = msg		
#pragma unroll 8
		for (int i = 0; i<8; i++)
			p[i] = inpHash[i];

		t0 = 64; // ptr
		t1 = 480ull << 55; // etype
		TFBIG_KINIT(h0, h1, h2, h3, h4, h5, h6, h7, h8, t0, t1, t2);
		TFBIG_4e(0);
		TFBIG_4o(1);
		TFBIG_4e(2);
		TFBIG_4o(3);
		TFBIG_4e(4);
		TFBIG_4o(5);
		TFBIG_4e(6);
		TFBIG_4o(7);
		TFBIG_4e(8);
		TFBIG_4o(9);
		TFBIG_4e(10);
		TFBIG_4o(11);
		TFBIG_4e(12);
		TFBIG_4o(13);
		TFBIG_4e(14);
		TFBIG_4o(15);
		TFBIG_4e(16);
		TFBIG_4o(17);
		TFBIG_ADDKEY(p[0], p[1], p[2], p[3], p[4], p[5], p[6], p[7], h, t, 18);

		h0 = inpHash[0] ^ p[0];
		h1 = inpHash[1] ^ p[1];
		h2 = inpHash[2] ^ p[2];
		h3 = inpHash[3] ^ p[3];
		h4 = inpHash[4] ^ p[4];
		h5 = inpHash[5] ^ p[5];
		h6 = inpHash[6] ^ p[6];
		h7 = inpHash[7] ^ p[7];

		// 2. Runde -> etype = 510, ptr = 8, bcount = 0, data = 0
#pragma unroll 8
		for (int i = 0; i<8; i++)
			p[i] = 0;

		t0 = 8; // ptr
		t1 = 510ull << 55; // etype
		TFBIG_KINIT(h0, h1, h2, h3, h4, h5, h6, h7, h8, t0, t1, t2);
		TFBIG_4e(0);
		TFBIG_4o(1);
		TFBIG_4e(2);
		TFBIG_4o(3);
		TFBIG_4e(4);
		TFBIG_4o(5);
		TFBIG_4e(6);
		TFBIG_4o(7);
		TFBIG_4e(8);
		TFBIG_4o(9);
		TFBIG_4e(10);
		TFBIG_4o(11);
		TFBIG_4e(12);
		TFBIG_4o(13);
		TFBIG_4e(14);
		TFBIG_4o(15);
		TFBIG_4e(16);
		TFBIG_4o(17);
		TFBIG_ADDKEY(p[0], p[1], p[2], p[3], p[4], p[5], p[6], p[7], h, t, 18);

		uint32_t *Hash = (uint32_t *)&p[0];	

		if (cuda_hashisbelowtarget(Hash, pTarget)) d_nonce[0] = nounce;
	}
}

__host__ void quark_skein512_cpu_init(int thr_id)
{
	hipMalloc(&d_nonce[thr_id], sizeof(uint32_t));
}

__host__ void quark_skein512_setTarget(const void *ptarget)
{
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(pTarget), ptarget, 8 * sizeof(uint32_t), 0, hipMemcpyHostToDevice));
}
__host__ void quark_skein512_cpu_free(int32_t thr_id)
{
	hipFree(pTarget);
	hipHostFree(&d_nonce[thr_id]);
}

__host__
void quark_skein512_cpu_hash_64(int thr_id, int threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order)
{
	const int threadsperblock = 448;

	// berechne wie viele Thread Blocks wir brauchen
	dim3 grid((threads + threadsperblock-1)/threadsperblock);
	dim3 block(threadsperblock);

	// Gr��e des dynamischen Shared Memory Bereichs
	size_t shared_size = 0;

	quark_skein512_gpu_hash_64 << <grid, block, shared_size >> >(threads, startNounce, (uint64_t*)d_hash, d_nonceVector);
	uint32_t res;
	hipMemcpy(&res, d_nonce[thr_id], sizeof(uint32_t), hipMemcpyDeviceToHost);
}

__host__
uint32_t quark_skein512_cpu_hash_64_final(int thr_id, int threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order)
{
	const int threadsperblock = 448;

	dim3 grid((threads + threadsperblock - 1) / threadsperblock);
	dim3 block(threadsperblock);

	size_t shared_size = 0;
	hipMemset(d_nonce[thr_id], 0xffffffff, sizeof(uint32_t));

	quark_skein512_gpu_hash_64_final<< <grid, block, shared_size >> >(threads, startNounce, (uint64_t*)d_hash, d_nonceVector, d_nonce[thr_id]);
	MyStreamSynchronize(NULL, order, thr_id);
	uint32_t res;
	hipMemcpy(&res, d_nonce[thr_id], sizeof(uint32_t), hipMemcpyDeviceToHost);
	return res;
}
