#include "hip/hip_runtime.h"
#include <stdio.h>
#include <memory.h>

#include "cuda_helper.h"

// Hash Target gegen das wir testen sollen
__constant__ uint32_t pTarget[8];

static uint32_t *d_resNounce[8];
static uint32_t *h_resNounce[8];

__global__
void cuda_check_gpu_hash_64(int threads, uint32_t startNounce, uint32_t *g_nonceVector, uint32_t *g_hash, uint32_t *resNounce)
{
	int thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		// bestimme den aktuellen Zähler
		uint32_t nounce = (g_nonceVector != NULL) ? g_nonceVector[thread] : (startNounce + thread);

		uint32_t hashPosition = (nounce - startNounce) << 4;
		uint32_t *inpHash = &g_hash[hashPosition];
		uint32_t hash[8];

		#pragma unroll 8
		for (int i=0; i < 8; i++)
			hash[i] = inpHash[i];

		for (int i = 7; i >= 0; i--) {
			if (hash[i] > pTarget[i]) {
				return;
			}
			if (hash[i] < pTarget[i]) {
				break;
			}
		}
		if (resNounce[0] > nounce)
			resNounce[0] = nounce;
	}
}

// Setup-Funktionen
__host__
void cuda_check_cpu_init(int thr_id, int threads)
{
    CUDA_CALL_OR_RET(hipHostMalloc(&h_resNounce[thr_id], 1*sizeof(uint32_t)));
    CUDA_CALL_OR_RET(hipMalloc(&d_resNounce[thr_id], 1*sizeof(uint32_t)));
}

// Target Difficulty
__host__
void cuda_check_cpu_setTarget(const void *ptarget)
{
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(pTarget), ptarget, 8*sizeof(uint32_t), 0, hipMemcpyHostToDevice));
}

__host__
uint32_t cuda_check_cpu_hash_64(int thr_id, int threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_inputHash, int order)
{
	uint32_t result = 0xffffffff;
	hipMemset(d_resNounce[thr_id], 0xff, sizeof(uint32_t));

	const int threadsperblock = 256;

	dim3 grid((threads + threadsperblock-1)/threadsperblock);
	dim3 block(threadsperblock);

	cuda_check_gpu_hash_64 <<<grid, block>>> (threads, startNounce, d_nonceVector, d_inputHash, d_resNounce[thr_id]);

	// Strategisches Sleep Kommando zur Senkung der CPU Last
	MyStreamSynchronize(NULL, order, thr_id);

	// Ergebnis zum Host kopieren (in page locked memory, damits schneller geht)
	hipMemcpy(h_resNounce[thr_id], d_resNounce[thr_id], sizeof(uint32_t), hipMemcpyDeviceToHost);

	// hipMemcpy() ist asynchron!
	hipDeviceSynchronize();
	result = *h_resNounce[thr_id];

	return result;
}

__global__
void cuda_check_gpu_hash_fast(int threads, uint32_t startNounce, uint32_t *hashEnd, uint32_t *resNounce)
{
	int thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		/* only test the last 2 dwords, ok for most algos */
		int hashPos = thread << 4;
		uint32_t *inpHash = &hashEnd[hashPos];

		if (inpHash[7] <= pTarget[7] && inpHash[6] <= pTarget[6]) {
			uint32_t nounce = (startNounce + thread);
			if (resNounce[0] > nounce)
				resNounce[0] = nounce;
		}
	}
}

__host__
uint32_t cuda_check_hash_fast(int thr_id, int threads, uint32_t startNounce, uint32_t *d_inputHash, int order)
{
	uint32_t result = 0xffffffff;
	hipMemset(d_resNounce[thr_id], 0xff, sizeof(uint32_t));

	const int threadsperblock = 256;

	dim3 grid((threads + threadsperblock - 1) / threadsperblock);
	dim3 block(threadsperblock);

	cuda_check_gpu_hash_fast <<<grid, block>>> (threads, startNounce, d_inputHash, d_resNounce[thr_id]);

	// MyStreamSynchronize(NULL, order, thr_id);
	hipDeviceSynchronize();

	hipMemcpy(h_resNounce[thr_id], d_resNounce[thr_id], sizeof(uint32_t), hipMemcpyDeviceToHost);

	// hipMemcpy() was asynchron ?
	// hipDeviceSynchronize();
	result = *h_resNounce[thr_id];

	return result;
}
