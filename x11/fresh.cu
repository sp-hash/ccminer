#include "hip/hip_runtime.h"
/**
 * Fresh algorithm
 */
extern "C" {
#include "sph/sph_shavite.h"
#include "sph/sph_simd.h"
#include "sph/sph_echo.h"
}
#include "miner.h"
#include "cuda_helper.h"

// to test gpu hash on a null buffer
#define NULLTEST 0

static uint32_t *d_hash[8];

extern void x11_shavite512_setBlock_80(void *pdata);
extern void x11_shavite512_cpu_hash_80(int thr_id, int threads, uint32_t startNounce, uint32_t *d_hash, int order);
extern void x11_shavite512_cpu_hash_64(int thr_id, int threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);

extern int  x11_simd512_cpu_init(int thr_id, int threads);
extern void x11_simd512_cpu_hash_64(int thr_id, int threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);

extern void x11_echo512_cpu_init(int thr_id, int threads);
extern void x11_echo512_cpu_hash_64(int thr_id, int threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);

extern void quark_compactTest_cpu_init(int thr_id, int threads);
extern void quark_compactTest_cpu_hash_64(int thr_id, int threads, uint32_t startNounce, uint32_t *inpHashes,
											uint32_t *d_noncesTrue, size_t *nrmTrue, uint32_t *d_noncesFalse, size_t *nrmFalse,
											int order);

// CPU Hash
extern "C" void fresh_hash(void *state, const void *input)
{
	// shavite-simd-shavite-simd-echo

	sph_shavite512_context ctx_shavite;
	sph_simd512_context ctx_simd;
	sph_echo512_context ctx_echo;

	unsigned char hash[128]; // uint32_t hashA[16], hashB[16];
	#define hashA hash
	#define hashB hash+64

	memset(hash, 0, sizeof hash);

	sph_shavite512_init(&ctx_shavite);
	sph_shavite512(&ctx_shavite, input, 80);
	sph_shavite512_close(&ctx_shavite, hashA);

	sph_simd512_init(&ctx_simd);
	sph_simd512(&ctx_simd, hashA, 64);
	sph_simd512_close(&ctx_simd, hashB);

	sph_shavite512_init(&ctx_shavite);
	sph_shavite512(&ctx_shavite, hashB, 64);
	sph_shavite512_close(&ctx_shavite, hashA);

	sph_simd512_init(&ctx_simd);
	sph_simd512(&ctx_simd, hashA, 64);
	sph_simd512_close(&ctx_simd, hashB);

	sph_echo512_init(&ctx_echo);
	sph_echo512(&ctx_echo, hashB, 64);
	sph_echo512_close(&ctx_echo, hashA);

	memcpy(state, hash, 32);
}

extern "C" int scanhash_fresh(int thr_id, uint32_t *pdata,
	const uint32_t *ptarget, uint32_t max_nonce,
	unsigned long *hashes_done)
{
	const uint32_t first_nonce = pdata[19];
	static bool init[8] = {0,0,0,0,0,0,0,0};
	uint32_t endiandata[20];

	int throughput = opt_work_size ? opt_work_size : (1 << 19); // 256*256*8;
	throughput = min(throughput, (int) (max_nonce - first_nonce));

	if (opt_benchmark)
		((uint32_t*)ptarget)[7] = 0x00ff;

	if (!init[thr_id])
	{
		hipSetDevice(device_map[thr_id]);

		x11_simd512_cpu_init(thr_id, throughput);
		x11_echo512_cpu_init(thr_id, throughput);

		CUDA_CALL_OR_RET_X(hipMalloc(&d_hash[thr_id], 16 * sizeof(uint32_t) * throughput + 4), 0);

		cuda_check_cpu_init(thr_id, throughput);

		init[thr_id] = true;
	}

	for (int k=0; k < 20; k++)
		be32enc(&endiandata[k], ((uint32_t*)pdata)[k]);
	
	x11_shavite512_setBlock_80((void*)endiandata);
	cuda_check_cpu_setTarget(ptarget);
	do {
		uint32_t Htarg = ptarget[7];

		uint32_t foundNonce;
		int order = 0;

		// GPU Hash
		x11_shavite512_cpu_hash_80(thr_id, throughput, pdata[19], d_hash[thr_id], order++);
		x11_simd512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		x11_shavite512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		x11_simd512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		x11_echo512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);

#if NULLTEST
		uint32_t buf[8]; memset(buf, 0, sizeof buf);
		CUDA_SAFE_CALL(hipMemcpy(buf, d_hash[thr_id], sizeof buf, hipMemcpyDeviceToHost));
		CUDA_SAFE_CALL(hipDeviceSynchronize());
		print_hash((unsigned char*)buf); printf("\n");
#endif

		foundNonce = cuda_check_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		if (foundNonce != 0xffffffff)
		{
			uint32_t vhash64[8];
			be32enc(&endiandata[19], foundNonce);
			fresh_hash(vhash64, endiandata);

			if (vhash64[7] <= Htarg && fulltest(vhash64, ptarget)) {
				*hashes_done = pdata[19] + throughput - first_nonce;
				pdata[19] = foundNonce;
				return 1;
			}
			else if (vhash64[7] > Htarg) {
				applog(LOG_INFO, "GPU #%d: result for %08x is not in range: %x > %x", thr_id, foundNonce, vhash64[7], Htarg);
			}
			else {
				applog(LOG_INFO, "GPU #%d: result for %08x does not validate on CPU!", thr_id, foundNonce);
			}
		}

		if (pdata[19] + throughput < pdata[19])
			pdata[19] = max_nonce;
		else pdata[19] += throughput;

	} while (pdata[19] < max_nonce && !work_restart[thr_id].restart);

	*hashes_done = pdata[19] - first_nonce + 1;
	return 0;
}
