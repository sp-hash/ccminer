extern "C"
{
#include "sph/sph_blake.h"
#include "sph/sph_groestl.h"
#include "sph/sph_skein.h"
#include "sph/sph_jh.h"
#include "sph/sph_keccak.h"
}

#include "miner.h"

#include "cuda_helper.h"

// Speicher für Input/Output der verketteten Hashfunktionen
static uint32_t *d_hash[8];

extern void quark_blake512_cpu_setBlock_80(void *pdata);
extern void quark_blake512_cpu_hash_80(int thr_id, int threads, uint32_t startNounce, uint32_t *d_hash, int order);

extern void quark_groestl512_cpu_init(int thr_id, int threads);
extern void quark_groestl512_cpu_hash_64(int thr_id, int threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);

//extern void quark_jh512_cpu_hash_64(int thr_id, int threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);
//extern void quark_keccak512_cpu_hash_64(int thr_id, int threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);
extern void cuda_jh512Keccak512_cpu_hash_64(int thr_id, int threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);

extern uint32_t quark_skein512_cpu_hash_64_final(int thr_id, int threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);
extern void quark_skein512_cpu_init(int thr_id);
extern void quark_skein512_setTarget(const void *ptarget);
extern void quark_skein512_cpu_free(int32_t thr_id);


// Original nist5hash Funktion aus einem miner Quelltext
extern "C" void nist5hash(void *state, const void *input)
{
    sph_blake512_context ctx_blake;
    sph_groestl512_context ctx_groestl;
    sph_jh512_context ctx_jh;
    sph_keccak512_context ctx_keccak;
    sph_skein512_context ctx_skein;
    
    uint8_t hash[64];

    sph_blake512_init(&ctx_blake);
    sph_blake512 (&ctx_blake, input, 80);
    sph_blake512_close(&ctx_blake, (void*) hash);
    
    sph_groestl512_init(&ctx_groestl);
    sph_groestl512 (&ctx_groestl, (const void*) hash, 64);
    sph_groestl512_close(&ctx_groestl, (void*) hash);

    sph_jh512_init(&ctx_jh);
    sph_jh512 (&ctx_jh, (const void*) hash, 64);
    sph_jh512_close(&ctx_jh, (void*) hash);

    sph_keccak512_init(&ctx_keccak);
    sph_keccak512 (&ctx_keccak, (const void*) hash, 64);
    sph_keccak512_close(&ctx_keccak, (void*) hash);

    sph_skein512_init(&ctx_skein);
    sph_skein512 (&ctx_skein, (const void*) hash, 64);
    sph_skein512_close(&ctx_skein, (void*) hash);

    memcpy(state, hash, 32);
}

extern "C" int scanhash_nist5(int thr_id, uint32_t *pdata,
    const uint32_t *ptarget, uint32_t max_nonce,
    unsigned long *hashes_done)
{
	const uint32_t first_nonce = pdata[19];

	if (opt_benchmark)
		((uint32_t*)ptarget)[7] = 0x5;

	int intensity = (device_sm[device_map[thr_id]] > 500) ? 256*4096 * 12 : 256 * 4096 * 10;
	int throughput = opt_work_size ? opt_work_size : intensity; // 20=256*256*16;

	static bool init[8] = {0,0,0,0,0,0,0,0};
	if (!init[thr_id])
	{
		hipSetDevice(device_map[thr_id]);

		// Konstanten kopieren, Speicher belegen
		hipMalloc(&d_hash[thr_id], 16 * sizeof(uint32_t) * throughput);
		quark_groestl512_cpu_init(thr_id, throughput);
		quark_skein512_cpu_init(thr_id);
		init[thr_id] = true;
	}

	uint32_t endiandata[20];
	for (int k=0; k < 20; k++)
		be32enc(&endiandata[k], ((uint32_t*)pdata)[k]);

	quark_blake512_cpu_setBlock_80((void*)endiandata);
	quark_skein512_setTarget(ptarget);

	do {
		int order = 0;

		// Hash with CUDA
		quark_blake512_cpu_hash_80(thr_id, throughput, pdata[19], d_hash[thr_id], order++);
		quark_groestl512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		cuda_jh512Keccak512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		uint32_t foundNonce = quark_skein512_cpu_hash_64_final(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);

		if  (foundNonce != 0xffffffff)
		{
			uint32_t vhash64[8];
			uint32_t Htarg = ptarget[7];
			be32enc(&endiandata[19], foundNonce);
			nist5hash(vhash64, endiandata);

			if ((vhash64[7]<=Htarg) && fulltest(vhash64, ptarget)) {

				*hashes_done = pdata[19] + throughput - first_nonce;
				pdata[19] = foundNonce;
				quark_skein512_cpu_free(thr_id);
				if (opt_benchmark) applog(LOG_INFO, "Found nounce", thr_id, foundNonce, vhash64[7], Htarg);
				return 1;
			} else {
				applog(LOG_INFO, "GPU #%d: result for nonce $%08X does not validate on CPU!", thr_id, foundNonce);
			}
		}
		if (pdata[19] + throughput < pdata[19])
			pdata[19] = max_nonce;
		else pdata[19] += throughput;

	} while (pdata[19] < max_nonce && !work_restart[thr_id].restart);

	*hashes_done = pdata[19] - first_nonce + 1;
	return 0;
}
