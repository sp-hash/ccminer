/*
 * whirlpool routine (djm)
 */
extern "C"
{
#include "sph/sph_whirlpool.h"
#include "miner.h"
}

// from cpu-miner.c
extern int device_map[8];

// Speicher für Input/Output der verketteten Hashfunktionen
static uint32_t *d_hash[8];

extern void x15_whirlpool_cpu_init(int thr_id, int threads, int mode);
extern void x15_whirlpool_cpu_hash_64(int thr_id, int threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);

extern void whirlpool512_setBlock_80(void *pdata, const void *ptarget);
extern void whirlpool512_cpu_hash_80(int thr_id, int threads, uint32_t startNounce, uint32_t *d_hash, int order);
extern uint32_t whirlpool512_cpu_finalhash_64(int thr_id, int threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);


// CPU Hash function
extern "C" void wcoinhash(void *state, const void *input)
{
	sph_whirlpool_context ctx_whirlpool;

	unsigned char hash[128]; // uint32_t hashA[16], hashB[16];
	#define hashB hash+64

	memset(hash, 0, sizeof hash);

	// shavite 1
	sph_whirlpool1_init(&ctx_whirlpool);
	sph_whirlpool1(&ctx_whirlpool, input, 80);
	sph_whirlpool1_close(&ctx_whirlpool, hash);

	sph_whirlpool1_init(&ctx_whirlpool);
	sph_whirlpool1(&ctx_whirlpool, hash, 64);
	sph_whirlpool1_close(&ctx_whirlpool, hashB);

	sph_whirlpool1_init(&ctx_whirlpool);
	sph_whirlpool1(&ctx_whirlpool, hashB, 64);
	sph_whirlpool1_close(&ctx_whirlpool, hash);

	sph_whirlpool1_init(&ctx_whirlpool);
	sph_whirlpool1(&ctx_whirlpool, hash, 64);
	sph_whirlpool1_close(&ctx_whirlpool, hash);

	memcpy(state, hash, 32);
}

extern "C" int scanhash_whc(int thr_id, uint32_t *pdata,
    const uint32_t *ptarget, uint32_t max_nonce,
    unsigned long *hashes_done)
{
	const uint32_t first_nonce = pdata[19];
	const int throughput = 256*256*8;
	static bool init[8] = {0,0,0,0,0,0,0,0};
	uint32_t endiandata[20];
	uint32_t Htarg = ptarget[7];

	if (opt_benchmark)
		((uint32_t*)ptarget)[7] = Htarg = 0x0000ff;

	if (!init[thr_id]) {
		hipSetDevice(device_map[thr_id]);
		// Konstanten kopieren, Speicher belegen
		hipMalloc(&d_hash[thr_id], 16 * sizeof(uint32_t) * throughput);
		x15_whirlpool_cpu_init(thr_id, throughput, 1 /* old whirlpool */);

		init[thr_id] = true;
	}

	for (int k=0; k < 20; k++) {
		be32enc(&endiandata[k], ((uint32_t*)pdata)[k]);
	}

	whirlpool512_setBlock_80((void*)endiandata, ptarget);

	do {
		uint32_t foundNonce;
		int order = 0;

		whirlpool512_cpu_hash_80(thr_id, throughput, pdata[19], d_hash[thr_id], order++);
		x15_whirlpool_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		x15_whirlpool_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);

		foundNonce = whirlpool512_cpu_finalhash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		if (foundNonce != 0xffffffff)
		{
			uint32_t vhash64[8];
			be32enc(&endiandata[19], foundNonce);

			wcoinhash(vhash64, endiandata);

			if (vhash64[7] <= Htarg && fulltest(vhash64, ptarget))
			{
				*hashes_done = pdata[19] + throughput - first_nonce;
				pdata[19] = foundNonce;
				return 1;
			}
			else if (vhash64[7] > Htarg) {
				applog(LOG_INFO, "GPU #%d: result for %08x is not in range: %x > %x", thr_id, foundNonce, vhash64[7], Htarg);
			}
			else {
				applog(LOG_INFO, "GPU #%d: result for %08x does not validate on CPU!", thr_id, foundNonce);
			}
		}
		if (pdata[19] + throughput < pdata[19])
			pdata[19] = max_nonce;
		else pdata[19] += throughput;

	} while (pdata[19] < max_nonce && !work_restart[thr_id].restart);

	*hashes_done = pdata[19] - first_nonce + 1;
	return 0;
}
