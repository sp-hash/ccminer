#include "hip/hip_runtime.h"
/*
 * deepcoin algorithm
 *
 */
extern "C" {
#include "sph/sph_luffa.h"
#include "sph/sph_cubehash.h"
#include "sph/sph_shavite.h"
#include "sph/sph_simd.h"
#include "sph/sph_echo.h"
}

#include "miner.h"

#include "cuda_helper.h"

static uint32_t *d_hash[8];

extern void qubit_luffa512_cpu_init(int thr_id, int threads);
extern void qubit_luffa512_cpu_setBlock_80(void *pdata);
extern void qubit_luffa512_cpu_hash_80(int thr_id, int threads, uint32_t startNounce, uint32_t *d_hash, int order);
extern void qubit_luffa512_cpufinal_setBlock_80(void *pdata, const void *ptarget);
extern uint32_t qubit_luffa512_cpu_finalhash_80(int thr_id, int threads, uint32_t startNounce, uint32_t *d_hash, int order);

extern void x11_cubehash512_cpu_hash_64(int thr_id, int threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);

extern void x11_echo512_cpu_init(int thr_id, int threads);
extern void x11_echo512_cpu_hash_64(int thr_id, int threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order);

extern "C" void deephash(void *state, const void *input)
{
	// luffa1-cubehash2-shavite3-simd4-echo5
	sph_luffa512_context ctx_luffa;
	sph_cubehash512_context ctx_cubehash;
	sph_echo512_context ctx_echo;

	uint8_t hash[64];

	sph_luffa512_init(&ctx_luffa);
	sph_luffa512 (&ctx_luffa, input, 80);
	sph_luffa512_close(&ctx_luffa, (void*) hash);

	sph_cubehash512_init(&ctx_cubehash);
	sph_cubehash512 (&ctx_cubehash, (const void*) hash, 64);
	sph_cubehash512_close(&ctx_cubehash, (void*) hash);

	sph_echo512_init(&ctx_echo);
	sph_echo512 (&ctx_echo, (const void*) hash, 64);
	sph_echo512_close(&ctx_echo, (void*) hash);

	memcpy(state, hash, 32);
}


extern "C" int scanhash_deep(int thr_id, uint32_t *pdata,
	const uint32_t *ptarget, uint32_t max_nonce,
	unsigned long *hashes_done)
{
	const uint32_t first_nonce = pdata[19];
	static bool init[8] = {0,0,0,0,0,0,0,0};
	uint32_t endiandata[20];
	int throughput = opt_work_size ? opt_work_size : (1 << 19); // 256*256*8
	throughput = min(throughput, (int)(max_nonce - first_nonce));

	if (opt_benchmark)
		((uint32_t*)ptarget)[7] = 0x0000f;

	if (!init[thr_id])
	{
		hipSetDevice(device_map[thr_id]);
		hipMalloc(&d_hash[thr_id], 16 * sizeof(uint32_t) * throughput);

		qubit_luffa512_cpu_init(thr_id, throughput);
		x11_echo512_cpu_init(thr_id, throughput);

		cuda_check_cpu_init(thr_id, throughput);

		init[thr_id] = true;
	}

	for (int k=0; k < 20; k++)
		be32enc(&endiandata[k], ((uint32_t*)pdata)[k]);

	qubit_luffa512_cpufinal_setBlock_80((void*)endiandata,ptarget);
	cuda_check_cpu_setTarget(ptarget);

	do {
		const uint32_t Htarg = ptarget[7];
		int order = 0;

		qubit_luffa512_cpu_hash_80(thr_id, throughput, pdata[19], d_hash[thr_id], order++);
		x11_cubehash512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		x11_echo512_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		uint32_t foundNonce = cuda_check_cpu_hash_64(thr_id, throughput, pdata[19], NULL, d_hash[thr_id], order++);
		if (foundNonce != 0xffffffff)
		{
			uint32_t vhash64[8];
			be32enc(&endiandata[19], foundNonce);
			deephash(vhash64, endiandata);

			if (vhash64[7]<=Htarg && fulltest(vhash64, ptarget) )
			{
				*hashes_done = pdata[19] + throughput - first_nonce;
				pdata[19] = foundNonce;
				return 1;
			} else {
				applog(LOG_INFO, "GPU #%d: result for nonce $%08X does not validate on CPU!", thr_id, foundNonce);
			}
		}

		if (pdata[19] + throughput < pdata[19])
			pdata[19] = max_nonce;
		else pdata[19] += throughput;

	} while (pdata[19] < max_nonce && !work_restart[thr_id].restart);

	*hashes_done = pdata[19] - first_nonce + 1;
	return 0;
}
