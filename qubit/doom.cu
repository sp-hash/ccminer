#include "hip/hip_runtime.h"
/*
 * qubit algorithm
 *
 */
extern "C" {
#include "sph/sph_luffa.h"
}

#include "miner.h"

#include "cuda_helper.h"

static uint32_t *d_hash[8];

extern void qubit_luffa512_cpu_init(int thr_id, int threads);
extern void qubit_luffa512_cpu_setBlock_80(void *pdata);
extern void qubit_luffa512_cpu_hash_80(int thr_id, int threads, uint32_t startNounce, uint32_t *d_hash, int order);
extern void qubit_luffa512_cpufinal_setBlock_80(void *pdata, const void *ptarget);
extern uint32_t qubit_luffa512_cpu_finalhash_80(int thr_id, int threads, uint32_t startNounce, uint32_t *d_hash, int order);

extern void doomhash(void *state, const void *input)
{
	// luffa512
	sph_luffa512_context ctx_luffa;

	uint8_t hash[64];

	sph_luffa512_init(&ctx_luffa);
	sph_luffa512 (&ctx_luffa, input, 80);
	sph_luffa512_close(&ctx_luffa, (void*) hash);

	memcpy(state, hash, 32);
}


extern "C" int scanhash_doom(int thr_id, uint32_t *pdata,
	const uint32_t *ptarget, uint32_t max_nonce,
	unsigned long *hashes_done)
{
	const uint32_t first_nonce = pdata[19];
	static bool init[8] = {0,0,0,0,0,0,0,0};
	uint32_t endiandata[20];
	int throughput = opt_work_size ? opt_work_size : (1 << 22); // 256*256*8*8
	throughput = min(throughput, (int)(max_nonce - first_nonce));

	if (opt_benchmark)
		((uint32_t*)ptarget)[7] = 0x0000f;

	if (!init[thr_id])
	{
		hipSetDevice(device_map[thr_id]);

		CUDA_SAFE_CALL(hipMalloc(&d_hash[thr_id], 16 * sizeof(uint32_t) * throughput));

		qubit_luffa512_cpu_init(thr_id, throughput);

		init[thr_id] = true;
	}

	for (int k=0; k < 20; k++)
		be32enc(&endiandata[k], ((uint32_t*)pdata)[k]);

	qubit_luffa512_cpufinal_setBlock_80((void*)endiandata,ptarget);

	do {
		const uint32_t Htarg = ptarget[7];
		int order = 0;

		uint32_t foundNonce = qubit_luffa512_cpu_finalhash_80(thr_id, throughput, pdata[19], d_hash[thr_id], order++);
		if (foundNonce != 0xffffffff)
		{
			uint32_t vhash64[8];
			be32enc(&endiandata[19], foundNonce);
			doomhash(vhash64, endiandata);

			if (vhash64[7] <= Htarg && fulltest(vhash64, ptarget) )
			{
				*hashes_done = pdata[19] + throughput - first_nonce;
				pdata[19] = foundNonce;
				return 1;
			} else {
				applog(LOG_INFO, "GPU #%d: result for nonce $%08X does not validate on CPU!", thr_id, foundNonce);
			}
		}


		if ((uint64_t) pdata[19] + throughput > max_nonce) {
			pdata[19] = max_nonce;
			break;
		}
		pdata[19] += throughput;

	} while (!work_restart[thr_id].restart);

	*hashes_done = pdata[19] - first_nonce + 1;
	return 0;
}
